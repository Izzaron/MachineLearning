#include "hip/hip_runtime.h"
﻿#include "NumericalGPU.hpp"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

// This function is to be removed. It is merely kept as an example of error handling

namespace NumericalGPU {
    __global__ void addKernel(float* c, const float* a, const float* b)
    {
        int i = threadIdx.x;
        c[i] = a[i] - b[i];
    }

    hipError_t addOnDevice(float* c, const float* a, const float* b, float* dev_c, float* dev_a, float* dev_b, unsigned int size) {

        hipError_t cudaStatus;

        // Choose which GPU to run on, change this on a multi-GPU system.
        cudaStatus = hipSetDevice(0);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
            return cudaStatus;
        }

        // Allocate GPU buffers for three vectors (two input, one output)    .
        cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            return cudaStatus;
        }

        cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            return cudaStatus;
        }

        cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            return cudaStatus;
        }

        // Copy input vectors from host memory to GPU buffers.
        cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            return cudaStatus;
        }

        cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            return cudaStatus;
        }

        // Launch a kernel on the GPU with one thread for each element.
        addKernel << <1, size >> > (dev_c, dev_a, dev_b);

        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            return cudaStatus;
        }

        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
            return cudaStatus;
        }

        // Copy output vector from GPU buffer to host memory.
        cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            return cudaStatus;
        }
    }

    // Helper function for using CUDA to add vectors in parallel.
    int addWithCuda(float* c, const float* a, const float* b, unsigned int size)
    {
        float* dev_a = 0;
        float* dev_b = 0;
        float* dev_c = 0;

        //Add vectors in parallel.
        hipError_t cudaStatus = addOnDevice(c, a, b, dev_c, dev_a, dev_b, size);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "addWithCuda failed!");
            return 1;
        }

        hipFree(dev_c);
        hipFree(dev_a);
        hipFree(dev_b);

        // hipDeviceReset must be called before exiting in order for profiling and
        // tracing tools such as Nsight and Visual Profiler to show complete traces.
        cudaStatus = hipDeviceReset();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceReset failed!");
            return 1;
        }

        return 0;
    }
}
